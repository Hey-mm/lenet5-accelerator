
// modified by Yuxi Sun
// Keras trained accuracy 98.89%

// comment out to use gpu
#define CPU 

#include "parameters.h"
#include "lenet5.h"
#include "conv.h"

//hls-fpga-machine-learning insert weights
#include "../firmware/weights/w1.h"
#include "../firmware/weights/b1.h"
#include "../firmware/weights/w3.h"
#include "../firmware/weights/b3.h"
#include "../firmware/weights/w5.h"
#include "../firmware/weights/b5.h"
#include "../firmware/weights/w6.h"
#include "../firmware/weights/b6.h"
#include "../firmware/weights/w7.h"
#include "../firmware/weights/b7.h"


void lenet5(
		  input_t data[IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1],
		  result_t res[N_OUTPUTS])
{
    // ****************************************
    // NETWORK INSTANTIATION
    // ****************************************

    //hls-fpga-machine-learning insert layers

#ifndef CPU
    float *data_copy, *w1_copy, *b1_copy;
    hipMallocManaged(&data_copy, IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1 * sizeof(float));
    hipMallocManaged(&w1_copy, 150 * sizeof(float));
    hipMallocManaged(&b1_copy, 6 * sizeof(float));
    hipMemcpy(data_copy, data, sizeof(float)*IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1, hipMemcpyHostToDevice);
    hipMemcpy(w1_copy, w1, sizeof(float)*150, hipMemcpyHostToDevice);
    hipMemcpy(w1_copy, b1, sizeof(float)*6, hipMemcpyHostToDevice);

    float *conv2d_layer1_out;
    hipMallocManaged(&conv2d_layer1_out, OUT_HEIGHT_1*OUT_WIDTH_1*N_FILT_1 * sizeof(float));
    hipDeviceSynchronize();

    conv_2d_1<<<1,1>>>(data_copy, conv2d_layer1_out, w1_copy, b1_copy);
    hipDeviceSynchronize();
#else
    float conv2d_layer1_out[OUT_HEIGHT_1*OUT_WIDTH_1*N_FILT_1];
    conv_2d_1_cpu(data, conv2d_layer1_out, w1, b1);
#endif

    float pool2d_layer2_out[OUT_HEIGHT_2*OUT_WIDTH_2*N_FILT_2];
    nnet::pooling2d<config2>(conv2d_layer1_out, pool2d_layer2_out);

    float conv2d_layer3_out[OUT_HEIGHT_3 * OUT_WIDTH_3 * N_FILT_3];

    conv_2d_2(pool2d_layer2_out, conv2d_layer3_out, w3, b3);

    float layer4_out[OUT_HEIGHT_4*OUT_WIDTH_4*N_FILT_4];
    nnet::pooling2d<config4>(conv2d_layer3_out, layer4_out);

    float layer5_out[N_LAYER_5];
    nnet::compute_layer<config5>(layer4_out, layer5_out, w5, b5);

    float layer6_out[N_LAYER_6];
    nnet::compute_layer<config6>(layer5_out, layer6_out, w6, b6);

    // float logits7[N_OUTPUTS];

    nnet::compute_layer<config7>(layer6_out, res, w7, b7);

    // todo change to the non-table version of softmax
    // nnet::softmax<float, result_t, softmax_config7>(logits7, res); 

    // hipFree(data_copy);
    // hipFree(w1_copy);
    // hipFree(b1_copy);
    // hipFree(conv2d_layer1_out);
}
