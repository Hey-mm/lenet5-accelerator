#include "hip/hip_runtime.h"

// modified by Yuxi Sun
// Keras trained accuracy 98.89%

// comment out to use gpu
#define USE_CPU 

#include "parameters.h"
#include "lenet5.h"
// #include "conv.h"

//hls-fpga-machine-learning insert weights
#include "../firmware/weights/w1.h"
#include "../firmware/weights/b1.h"
#include "../firmware/weights/w3.h"
#include "../firmware/weights/b3.h"
#include "../firmware/weights/w5.h"
#include "../firmware/weights/b5.h"
#include "../firmware/weights/w6.h"
#include "../firmware/weights/b6.h"
#include "../firmware/weights/w7.h"
#include "../firmware/weights/b7.h"

__host__ void kernel_cpu(input_t data[IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1],
    result_t res[N_OUTPUTS])
{
    float conv2d_layer1_out[OUT_HEIGHT_1*OUT_WIDTH_1*N_FILT_1];
    nnet::conv_2d<config1>(data, conv2d_layer1_out, w1, b1);

    float pool2d_layer2_out[OUT_HEIGHT_2*OUT_WIDTH_2*N_FILT_2];
    nnet::pooling2d<config2>(conv2d_layer1_out, pool2d_layer2_out);

    float conv2d_layer3_out[OUT_HEIGHT_3 * OUT_WIDTH_3 * N_FILT_3];
    nnet::conv_2d<config3>(pool2d_layer2_out, conv2d_layer3_out, w3, b3);

    float layer4_out[OUT_HEIGHT_4*OUT_WIDTH_4*N_FILT_4];
    nnet::pooling2d<config4>(conv2d_layer3_out, layer4_out);

    float layer5_out[N_LAYER_5];
    nnet::compute_layer<config5>(layer4_out, layer5_out, w5, b5);

    float layer6_out[N_LAYER_6];
    nnet::compute_layer<config6>(layer5_out, layer6_out, w6, b6);

    // float logits7[N_OUTPUTS];

    nnet::compute_layer<config7>(layer6_out, res, w7, b7);

    // todo change to the non-table version of softmax
    // nnet::softmax<float, result_t, softmax_config7>(logits7, res); 
}

__global__ void kernel(input_t data[IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1],
    result_t res[N_OUTPUTS])
{
    float conv2d_layer1_out[OUT_HEIGHT_1*OUT_WIDTH_1*N_FILT_1];
    nnet::conv_2d<config1>(data, conv2d_layer1_out, w1, b1);

    float pool2d_layer2_out[OUT_HEIGHT_2*OUT_WIDTH_2*N_FILT_2];
    nnet::pooling2d<config2>(conv2d_layer1_out, pool2d_layer2_out);

    float conv2d_layer3_out[OUT_HEIGHT_3 * OUT_WIDTH_3 * N_FILT_3];
    nnet::conv_2d<config3>(pool2d_layer2_out, conv2d_layer3_out, w3, b3);

    float layer4_out[OUT_HEIGHT_4*OUT_WIDTH_4*N_FILT_4];
    nnet::pooling2d<config4>(conv2d_layer3_out, layer4_out);

    float layer5_out[N_LAYER_5];
    nnet::compute_layer<config5>(layer4_out, layer5_out, w5, b5);

    float layer6_out[N_LAYER_6];
    nnet::compute_layer<config6>(layer5_out, layer6_out, w6, b6);

    // float logits7[N_OUTPUTS];
    nnet::compute_layer<config7>(layer6_out, res, w7, b7);

    // todo change to the non-table version of softmax
    // nnet::softmax<float, result_t, softmax_config7>(logits7, res); 
}

void lenet5(input_t data[IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1],
		  result_t res[N_OUTPUTS])
{
    // ****************************************
    // NETWORK INSTANTIATION
    // ****************************************

    //hls-fpga-machine-learning insert layers

#ifndef USE_CPU
    // data
    float *data_copy; 
    hipMallocManaged(&data_copy, IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1 * sizeof(float));
    hipMemcpy(data_copy, data, sizeof(float)*IN_HEIGHT_1*IN_WIDTH_1*N_CHAN_1, hipMemcpyHostToDevice);

    // layer1
    float *w1_copy, *b1_copy;
    hipMallocManaged(&w1_copy, 150 * sizeof(float));
    hipMallocManaged(&b1_copy, 6 * sizeof(float));
    hipMemcpy(w1_copy, w1, sizeof(float)*150, hipMemcpyHostToDevice);
    hipMemcpy(b1_copy, b1, sizeof(float)*6, hipMemcpyHostToDevice);

    // layer 3
    float *w3_copy, *b3_copy;
    hipMallocManaged(&w3_copy, 2400 * sizeof(float));
    hipMallocManaged(&b3_copy, 16 * sizeof(float));
    hipMemcpy(w3_copy, w3, sizeof(float)*2400, hipMemcpyHostToDevice);
    hipMemcpy(b3_copy, b3, sizeof(float)*16, hipMemcpyHostToDevice);

    // layer 5
    float *w5_copy, *b5_copy;
    hipMallocManaged(&w5_copy, 30720 * sizeof(float));
    hipMallocManaged(&b5_copy, 120 * sizeof(float));
    hipMemcpy(w5_copy, w5, sizeof(float)*30720, hipMemcpyHostToDevice);
    hipMemcpy(b5_copy, b5, sizeof(float)*120, hipMemcpyHostToDevice);

    // layer 6
    float *w6_copy, *b6_copy;
    hipMallocManaged(&w6_copy, 10080 * sizeof(float));
    hipMallocManaged(&b6_copy, 84 * sizeof(float));
    hipMemcpy(w6_copy, w6, sizeof(float)*10080, hipMemcpyHostToDevice);
    hipMemcpy(b6_copy, b6, sizeof(float)*84, hipMemcpyHostToDevice);

    // layer 7
    float *w7_copy, *b7_copy;
    hipMallocManaged(&w7_copy, 840 * sizeof(float));
    hipMallocManaged(&b7_copy, 10 * sizeof(float));
    hipMemcpy(w7_copy, w7, sizeof(float)*840, hipMemcpyHostToDevice);
    hipMemcpy(b7_copy, b7, sizeof(float)*10, hipMemcpyHostToDevice);
    
    // result
    float *res_copy;
    hipMallocManaged(&res_copy, N_OUTPUTS * sizeof(float));

    // sync
    hipDeviceSynchronize();

    // int block_size_1 = 32;
    // int num_blocks_1 = (OUT_HEIGHT_1 + block_size_1 - 1)/block_size_1;
    kernel<<<1,1>>>(data_copy, res_copy);
    hipDeviceSynchronize();

    // copy back
    hipMemcpy(res, res_copy, sizeof(float)*N_OUTPUTS, hipMemcpyDeviceToHost);

    // clean up
    hipFree(data_copy);
    hipFree(w1_copy);
    hipFree(b1_copy);
    hipFree(w3_copy);
    hipFree(b3_copy);
    hipFree(w5_copy);
    hipFree(b5_copy);
    hipFree(w6_copy);
    hipFree(b6_copy);
    hipFree(w7_copy);
    hipFree(b7_copy);
#else
    kernel_cpu(data, res);
#endif
}
